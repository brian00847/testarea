

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add_integers(int* a, int* b, int* c)
{
  *c = *a + *b;
}

__global__ void mykernel(void) {
}

int main(void) {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    add_integers<<<1,1>>>(d_a, d_b, d_c);
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    
    printf("The answer is %d\n", c);
    return 0;
}
