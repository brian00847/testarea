

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512


__global__ void add_integers(int* a, int* b, int* c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* arr, const int n_elements)
{
    for ( int i = 0; i < n_elements; ++i)
    {
        arr[i] = i;
    }
}


int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);random_ints(c, N);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    add_integers<<<N,1>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    for ( int i = 0; i < N; ++i)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
