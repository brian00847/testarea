

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 2048 * 2048
#define THREADS_PER_BLOCK 512


__global__ void add_integers(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void random_ints(int* arr, const int n_elements)
{
    for ( int i = 0; i < n_elements; ++i)
    {
        arr[i] = i;
    }
}


int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size); random_ints(c, N);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    printf("Starting Kernel\n");
    add_integers<<<N/ THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    printf("Kernel Complete!\n");
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Iterating\n");
    for ( int i = 0; i < N; ++i)
    {
        c[i] = a[i] + b[i];
    }
    printf("Iterating Complete\n");
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
